#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Compute C = A*B

__global__ void matrixMultiply(float * A, float * B, float *C,
                    int numAColumns, int numBColumns,
                    int numCRows, int numCColumns) {
   int Row = blockIdx.y * blockDim.y + threadIdx.y;
   int Col = blockIdx.x * blockDim.x + threadIdx.x;
   if( (Row < numCRows) && (Col < numCColumns) ) {
      float Cvalue = 0.0;
      for(int i = 0; i < numAColumns; ++i) {
         Cvalue += A[Row*numAColumns + i]*B[i*numBColumns + Col];
      }
      C[Row*numCColumns + Col] = Cvalue;
   } // if( (Row < numCRows) && (Col < numCColumns) )
}

int main(int argc, char** argv) {
   float * hostA;  // The A matrix
   float * hostB;  // The B matrix
   float * hostC;  // The output C matrix
   float * hostCompare;
   float * deviceA;
   float * deviceB;
   float * deviceC;
   int numARows; // number of rows in the matrix A
   int numAColumns; // number of columns in the matrix A
   int numBRows; // number of rows in the matrix B
   int numBColumns; // number of columns in the matrix B
   int numCRows; // number of rows in the matrix C
   int numCColumns; // number of columns in the matrix C

   int i, j, k, offsetA, offsetB, offsetC;
   size_t num_bad_values;
   float sum, delta;

   numARows = 100;
   numAColumns = 50;
   numBRows = 50;
   numBColumns = 200;
   
   // Set numCRows and numCColumns
   numCRows = numARows;
   numCColumns = numBColumns;

   // Allocate host matrices.
   hostA = (float *) malloc(numARows * numAColumns * sizeof(float));
   hostB = (float *) malloc(numBRows * numBColumns * sizeof(float));
   hostC = (float *) malloc(numCRows * numCColumns * sizeof(float));
   hostCompare = (float *) malloc(numCRows * numCColumns * sizeof(float));

   // Fill-in values for hostA and hostB.
   for(i = 0; i < numARows; i++) {
     for(j = 0; j < numAColumns; j++) {
       offsetA = j + i*numAColumns;
       hostA[offsetA] = 2.0*i + 3.0*j;
     }
   }
   for(i = 0; i < numBRows; i++) {
     for(j = 0; j < numBColumns; j++) {
       offsetB = j + i*numBColumns;
       hostB[offsetB] = 5.0*i + 7.0*j;
     }
   }
   // Calculate matrix multiplication on CPU.
   for(i = 0; i < numCColumns; i++) {
     for(j = 0; j < numCRows; j++) {
       offsetC = i + j*numCColumns;
       sum = 0.0;
       for(k = 0; k < numAColumns; k++) {
	 offsetA = k + j*numAColumns;
	 offsetB = i + k*numBColumns;
	 sum += hostA[offsetA]*hostB[offsetB];
       }
       hostCompare[offsetC] = sum;
     }
   }
   
   // Allocate GPU memory.
   hipMalloc(&deviceA, numARows * numAColumns * sizeof(float));
   hipMalloc(&deviceB, numBRows * numBColumns * sizeof(float));
   hipMalloc(&deviceC, numCRows * numCColumns * sizeof(float));


   // Copy host memory to the GPU.
   hipMemcpy(deviceA, hostA,
              numARows * numAColumns * sizeof(float),
              hipMemcpyHostToDevice);
   hipMemcpy(deviceB, hostB,
              numBRows * numBColumns * sizeof(float),
              hipMemcpyHostToDevice);

   // Initialize the grid and block dimensions.
   // 16x16 is a typical value, it could be changed.
   dim3 threadsPerBlock(16, 16);
   dim3 numBlocks((numCColumns + threadsPerBlock.x - 1)/threadsPerBlock.x,
                  (numCRows + threadsPerBlock.y - 1)/threadsPerBlock.y);

   // Launch the kernel.
   matrixMultiply<<<numBlocks, threadsPerBlock>>>(deviceA, deviceB, deviceC,
                    numAColumns, numBColumns,
                    numCRows, numCColumns);

   hipDeviceSynchronize();

   // Copy the GPU memory back to the CPU .
   hipMemcpy(hostC, deviceC,
              numCRows * numCColumns * sizeof(float),
              hipMemcpyDeviceToHost);

   delta = 0.0000001;
   num_bad_values = 0;
   for(i = 0; i < numCColumns; i++) {
     for(j = 0; j < numCRows; j++) {
       offsetC = i + j*numCColumns;
       if(fabs(hostC[offsetC] - hostCompare[offsetC]) > delta) {
	 num_bad_values++;
       }
     }
   }

   fprintf(stdout," Number of bad values in C = %lu\n", num_bad_values);

   // Free the GPU memory.
   hipFree(deviceA);
   hipFree(deviceB);
   hipFree(deviceC);
   // Free the CPU memory.
   free(hostA);
   free(hostB);
   free(hostC);
   free(hostCompare);

   return 0;
}

