#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512

// Vector sum of C = A + B

__global__ void vecadd(double * A, double * B, double *C, int N) {
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   if(i<N) { C[i] = A[i] + B[i]; }
}

int main(int argc, char** argv) {
   double * hostA;
   double * hostB;
   double * hostC;
   double * hostCompare;
   double * deviceA;
   double * deviceB;
   double * deviceC;
   int N = 10000;
   int i;
   double delta;
   int num_bad_values;

   // Allocate host matrices.
   hostA = (double *) malloc(N * sizeof(double));
   hostB = (double *) malloc(N * sizeof(double));
   hostC = (double *) malloc(N * sizeof(double));
   hostCompare = (double *) malloc(N * sizeof(double));

   // Fill-in values for hostA and hostB.
   for(i = 0; i < N; i++) {
     hostA[i] = 1.0 + i;
     hostB[i] = 1.0/(1.0 + i);
   }
   // Calculate on host for comparison.
   for(i = 0; i < N; i++) {
     hostCompare[i] = hostA[i] + hostB[i];
   }
   
   // Allocate GPU memory.
   hipMalloc(&deviceA, N * sizeof(double));
   hipMalloc(&deviceB, N * sizeof(double));
   hipMalloc(&deviceC, N * sizeof(double));


   // Copy memory to the GPU.
   hipMemcpy(deviceA, hostA,
              N * sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(deviceB, hostB,
              N * sizeof(double),
              hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    dim3 dimGrid(((N - 1)/BLOCK_SIZE) + 1, 1, 1);

   // Launch the kernel.
   vecadd<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, N);

   hipDeviceSynchronize();

   // Copy the GPU memory back to the CPU .
   hipMemcpy(hostC, deviceC,
              N * sizeof(double),
              hipMemcpyDeviceToHost);

   delta = DBL_EPSILON;
   num_bad_values = 0;
   for(i = 0; i < N; i++) {
      if(fabs(hostC[i] - hostCompare[i]) > delta) {
	 num_bad_values++;
     }
   }

   fprintf(stdout," Number of bad values in C = %lu\n", num_bad_values);

   // Free the GPU memory.
   hipFree(deviceA);
   hipFree(deviceB);
   hipFree(deviceC);
   // Free the CPU memory.
   free(hostA);
   free(hostB);
   free(hostC);
   free(hostCompare);

   return 0;
}

